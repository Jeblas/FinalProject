#include "hip/hip_runtime.h"
//
// Created by jbm on 12/5/18.
//

#include <hip/driver_types.h>
#include "cuda_naive.h"
#include <stdio.h>

__global__ void transpose(Complex * A, Complex * A_transpose, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < h && j < w)
		A_transpose[j * w + i] = A[i * w + j];

}

__global__ void fft_row(Complex * input, Complex * output, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < h && j < w) {
//		printf("hellow from (%d,%d)\n", i, j);
		output[i * w + j].real = 0;
		output[i * w + j].imag = 0;

		for (int k = 0; k < w; k++) {
			float W_real = cos(2 * M_PI * j * k / w);
//			printf("W_real: %f\n", W_real);
			float W_imag = -sin(2 * M_PI * j * k / w);
			output[i * w + j].real += (W_real * input[i * w + k].real - W_imag * input[i * w + k].imag);
			output[i * w + j].imag += (W_real * input[i * w + k].imag + W_imag * input[i * w + k].real);
		}
//		printf("r: %f\n",output[i * w + j].real);

	}


}


void cuda_naive(Complex * input_image, Complex * output_transform, InputImage input_image_meta) {
	Complex * d_input_image = NULL;
	Complex * d_output_transform = NULL;
	Complex * d_input_image_transpose = NULL;

	int w = input_image_meta.get_width();
	int h = input_image_meta.get_height();

	hipMalloc((void**)&d_input_image, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_output_transform, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_input_image_transpose, sizeof(Complex)*w*h);

	hipMemcpy(d_input_image, input_image, sizeof(Complex)*w*h, hipMemcpyHostToDevice);

	int grid_height = ceil((float)h/THREADS_PER_BLOCK_SIDE);
	int grid_width = ceil((float)h/THREADS_PER_BLOCK_SIDE);

	fft_row<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image, d_output_transform, w, h);
	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	fft_row<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image_transpose, d_output_transform, w, h);

	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	hipMemcpy(output_transform, d_input_image_transpose, sizeof(Complex)*w*h,hipMemcpyDeviceToHost);
}
