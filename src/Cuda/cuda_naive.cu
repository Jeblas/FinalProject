#include "hip/hip_runtime.h"
//
// Created by jbm on 12/5/18.
//


#include "cuda_naive.h"


__global__ void transpose(Complex * A, Complex * A_transpose, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < h && j < w)
		A_transpose[j * w + i] = A[i * w + j];

}
__global__ void transpose(hipComplex * A, hipComplex * A_transpose, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < h && j < w)
		A_transpose[j * w + i] = A[i * w + j];

}
__global__ void idft_row(Complex * input, Complex * output, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;

	int this_element = i * w + j;


	if(i < h && j < w) {

		output[this_element].real = 0;
		output[this_element].imag = 0;

		for (int k = 0; k < w; k++) {
			double angle = 2 * M_PI * j * k / w;
			float W_real = cos(angle);

			float W_imag = sin(angle);
			int neighboring_element = i * w + k;
			output[this_element].real += (W_real * input[neighboring_element].real - W_imag * input[neighboring_element].imag);
			output[this_element].imag += (W_real * input[neighboring_element].imag + W_imag * input[neighboring_element].real);
		}
		output[this_element].real /= w;
		output[this_element].imag /= w;


	}
}

__global__ void dft_row(Complex * input, Complex * output, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;

	int this_element = i * w + j;


	if(i < h && j < w) {

		output[this_element].real = 0;
		output[this_element].imag = 0;

		for (int k = 0; k < w; k++) {
			double angle = 2 * M_PI * j * k / w;
			float W_real = cos(angle);

			float W_imag = -sin(angle);
			int neighboring_element = i * w + k;
			output[this_element].real += (W_real * input[neighboring_element].real - W_imag * input[neighboring_element].imag);
			output[this_element].imag += (W_real * input[neighboring_element].imag + W_imag * input[neighboring_element].real);
		}


	}


}

void cuda_2d_dft(Complex * input_image, Complex * output_transform, InputImage input_image_meta, FourierDirection dir) {
	Complex * d_input_image = NULL;
	Complex * d_output_transform = NULL;
	Complex * d_input_image_transpose = NULL;
	std::cout << dir << std::endl;
	int w = input_image_meta.get_width();
	int h = input_image_meta.get_height();

	hipMalloc((void**)&d_input_image, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_output_transform, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_input_image_transpose, sizeof(Complex)*w*h);

	hipMemcpy(d_input_image, input_image, sizeof(Complex)*w*h, hipMemcpyHostToDevice);

	int grid_height = ceil((float)h/THREADS_PER_BLOCK_SIDE);
	int grid_width = ceil((float)h/THREADS_PER_BLOCK_SIDE);

	if(dir == FORWARD) {
		dft_row << < dim3(grid_width, grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE) >> >
													(d_input_image, d_output_transform, w, h);
	} else if(dir == REVERSE) {
		idft_row << < dim3(grid_width, grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE) >> >
													 (d_input_image, d_output_transform, w, h);
	}

	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	if(dir == FORWARD) {
		dft_row << < dim3(grid_width, grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE) >> >
													(d_input_image_transpose, d_output_transform, w, h);
	}else if(dir == REVERSE) {
		idft_row << < dim3(grid_width, grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE) >> >
													 (d_input_image_transpose, d_output_transform, w, h);
	}
	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	hipMemcpy(output_transform, d_input_image_transpose, sizeof(Complex)*w*h,hipMemcpyDeviceToHost);
}

csr_complex_Mat generate_B_k_csr(int n) {
	csr_complex_Mat ret;
	ret.nnz = 2*n;
	ret.csrValA = (hipComplex *) malloc(sizeof(hipComplex)*ret.nnz);
	ret.csrColIndA = (int *) malloc(sizeof(int)*ret.nnz);
	ret.csrRowPtrA = (int *) malloc(sizeof(int)*(n + 1));
	for(int i = 0; i < ret.nnz; i+=2) {
		ret.csrValA[i] = make_hipComplex(1, 0);
	}
	for(int i = 1; i < ret.nnz/2; i+=2) {
		float arg = M_PI*(i-1)/n;
		ret.csrValA[i] = make_hipComplex(cos(arg),-sin(arg));
	}
	for(int i = ret.nnz/2 + 1; i < ret.nnz;i += 2) {
		float arg = M_PI*(i - 1 - ret.nnz/2)/n;
		ret.csrValA[i] = make_hipComplex(-cos(arg), sin(arg));
	}
	for(int i = 0; i < ret.nnz/2; i+=2) {
		ret.csrColIndA[i] = i/2;
		ret.csrColIndA[i + ret.nnz/2] = i/2;
	}
	for(int i = 1; i < ret.nnz/2; i+=2) {
		ret.csrColIndA[i] = i/2 + n/2;
		ret.csrColIndA[i + ret.nnz/2] = i/2 + n/2;
	}
	for(int i = 0; i < n; i ++) {
		ret.csrRowPtrA[i] = 2*i;
	}
	ret.csrRowPtrA[n] = ret.nnz;
	return ret;
}

csr_complex_Mat kronecker_I_B_k(int n, int k, csr_complex_Mat B_k) {
	csr_complex_Mat ret;

	int dim_I = n/k;
	ret.nnz = B_k.nnz*dim_I;
	ret.csrValA = (hipComplex *) malloc(sizeof(hipComplex)*ret.nnz);
	ret.csrColIndA = (int *) malloc(sizeof(int)*ret.nnz);
	ret.csrRowPtrA = (int *) malloc(sizeof(int)*(n + 1));
//	for(int i = 0; i < B_k.nnz; i++) {
//		for(int j =0; j < dim_I; j++){
//
//		}
//	}
	for(int i = 0; i < n; i++) {
		ret.csrRowPtrA[i] = 2*i;
	}
	ret.csrRowPtrA[n] = ret.nnz;

	for(int j = 0; j < dim_I; j++) {
		for(int i = 0; i < B_k.nnz; i++) {
			int index = i + j*B_k.nnz;
			int col = B_k.csrColIndA[i] + j*k;

			ret.csrColIndA[i + j*B_k.nnz] = B_k.csrColIndA[i] + j*k;
			ret.csrValA[i + j*B_k.nnz] = B_k.csrValA[i]; //might not be integer index
		}
	}
	return ret;
}

__global__ void bit_reversal(hipComplex * d_row, hipComplex *d_reversed, int n) {
	int i = threadIdx.x  + blockIdx.x  + blockDim.x;

	unsigned long x = i;
	if(i < n) {
		x = (((x & 0xaaaaaaaa) >> 1)|((x & 0x55555555) << 1));
		x = (((x & 0xcccccccc) >> 2)|((x & 0x33333333) << 2));
		x = (((x & 0xf0f0f0f0) >> 4)|((x & 0x0f0f0f0f) << 4));
		x = (((x & 0xff00ff00) >> 8)|((x & 0x00ff00ff) << 8));
		d_reversed[((x >> 16) | (x << 16))] = d_row[i];
	}
}


void ufft_row(hipComplex * d_row, int n ) {

	hipsparseMatDescr_t descrA = NULL;
	const hipComplex alpha = make_hipComplex(1.0, 0.0);
	const hipComplex beta = make_hipComplex(0.0, 0.0);
	hipsparseHandle_t cusparse_handle = NULL;
	hipStream_t stream = NULL;

	hipStreamCreate(&stream);
	hipsparseCreate(&cusparse_handle);
	hipsparseSetStream(cusparse_handle, stream);
	hipsparseCreateMatDescr(&descrA);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);

	hipComplex * d_Ikbk_csrValA = NULL;
	int *  d_Ikbk_csrRowPtrA = NULL;
	int * d_Ikbk_csrColIndA = NULL;

	hipComplex * d_to_reverse = NULL;

	hipMalloc((void**)&d_Ikbk_csrValA,sizeof(hipComplex)*n*2);
	hipMalloc((void**)&d_Ikbk_csrRowPtrA,sizeof(hipComplex)*(n + 1));
	hipMalloc((void**)&d_Ikbk_csrColIndA,sizeof(int)*n*2);


	hipComplex * d_dense_matrix = NULL;
	hipMalloc((void**)&d_dense_matrix, sizeof(hipComplex)*n*n);

	hipComplex dense_matrix[n][n];

	hipMalloc((void**)&d_to_reverse,sizeof(hipComplex)*n);
	hipComplex * out = (hipComplex *) malloc(sizeof(hipComplex)*n);
	int num_blocks = ceil((float) n/32);
//	hipMemcpy(d_to_reverse, d_row, sizeof(hipComplex)*n, hipMemcpyDeviceToDevice);
//

		for (int k = 2; k < n; k*=2) {
//			construct B_k
			csr_complex_Mat B_k = generate_B_k_csr(k);
			csr_complex_Mat I_kron_B_k = kronecker_I_B_k(n, k, B_k);
			hipMemcpy(d_Ikbk_csrValA, I_kron_B_k.csrValA, sizeof(hipComplex)*I_kron_B_k.nnz,hipMemcpyHostToDevice);
			hipMemcpy(d_Ikbk_csrRowPtrA, I_kron_B_k.csrRowPtrA, sizeof(int)*(n+1),hipMemcpyHostToDevice);
			hipMemcpy(d_Ikbk_csrColIndA, I_kron_B_k.csrColIndA, sizeof(int)*I_kron_B_k.nnz,hipMemcpyHostToDevice);

//			hipsparseCcsr2dense(cusparse_handle,
//								n,
//								n,
//								descrA,
//								d_Ikbk_csrValA,
//								d_Ikbk_csrRowPtrA,
//								d_Ikbk_csrColIndA,
//								d_dense_matrix,
//								n);
//
//			hipMemcpy(dense_matrix, d_dense_matrix, sizeof(hipComplex)*n*n, hipMemcpyDeviceToHost);
//			for(int i = 0; i < n; i++) {
//				for(int j =0; j < n; j++) {
//					std::cout << std::fixed << std::setprecision(1);
//					std::cout << "(" << dense_matrix[i][j].x << "," << dense_matrix[i][j].y << ") ";
//				}
//				std::cout << std::endl;
//			}
//			std::cout << std::endl;

			//compute x = I_kron_B_k*x
			hipsparseCcsrmv(cusparse_handle,
							HIPSPARSE_OPERATION_NON_TRANSPOSE,
							n,
							n,
							I_kron_B_k.nnz,
							&alpha,
							descrA,
							d_Ikbk_csrValA,
							d_Ikbk_csrRowPtrA,
							d_Ikbk_csrColIndA,
							d_row,
							&beta,
							d_row);
			hipStreamSynchronize(stream);

		}
			csr_complex_Mat B_k = generate_B_k_csr(n);
//			csr_complex_Mat I_kron_B_k = kronecker_I_B_k(n,n, B_k);
			hipMemcpy(d_Ikbk_csrValA, B_k.csrValA, sizeof(hipComplex)*B_k.nnz,hipMemcpyHostToDevice);
			hipMemcpy(d_Ikbk_csrRowPtrA, B_k.csrRowPtrA, sizeof(int)*(n+1),hipMemcpyHostToDevice);
			hipMemcpy(d_Ikbk_csrColIndA, B_k.csrColIndA, sizeof(int)*B_k.nnz,hipMemcpyHostToDevice);

//			hipsparseCcsr2dense(cusparse_handle,
//								n,
//								n,
//								descrA,
//								d_Ikbk_csrValA,
//								d_Ikbk_csrRowPtrA,
//								d_Ikbk_csrColIndA,
//								d_dense_matrix,
//								n);
//
//			hipMemcpy(dense_matrix, d_dense_matrix, sizeof(hipComplex)*n*n, hipMemcpyDeviceToHost);
//			for(int i = 0; i < n; i++) {
//				for(int j =0; j < n; j++) {
//					std::cout << std::fixed << std::setprecision(1);
//					std::cout << "(" << dense_matrix[i][j].x << "," << dense_matrix[i][j].y << ") ";
//				}
//				std::cout << std::endl;
//			}
//			std::cout << std::endl;
//			for(int i = 0; i < B_k.nnz; i++) {
//				std::cout << B_k.csrColIndA[i] << "," ;
//
//			}
//			std::cout << std::endl;
			//compute x = I_kron_B_k*x
			hipsparseCcsrmv(cusparse_handle,
							HIPSPARSE_OPERATION_TRANSPOSE,
							n,
							n,
							B_k.nnz,
							&alpha,
							descrA,
							d_Ikbk_csrValA,
							d_Ikbk_csrRowPtrA,
							d_Ikbk_csrColIndA,
							d_row,
							&beta,
							d_to_reverse);
		hipStreamSynchronize(stream);
//		bit_reversal<<<num_blocks,32>>>(d_to_reverse, d_row, n);

		hipMemcpy(d_row, d_to_reverse, sizeof(hipComplex)*n, hipMemcpyDeviceToDevice);
//		for(int i = 0; i < n; i++) {
//			std::cout << out[i].x << ", ";
//		}
//		std::cout << std::endl;

		hipFree(d_Ikbk_csrColIndA);
		hipFree(d_Ikbk_csrRowPtrA);
		hipFree(d_Ikbk_csrValA);
		hipsparseDestroy(cusparse_handle);
		hipStreamDestroy(stream);
		hipsparseDestroyMatDescr(descrA);
}

void cuda_ufft(Complex *input_image, Complex * output_transform, InputImage input_image_meta) {
	hipComplex * d_input_image = NULL;
	hipComplex * d_output_transform = NULL;
	hipComplex * d_input_image_transpose = NULL;

	int w = input_image_meta.get_width();
	int h = input_image_meta.get_height();

	hipComplex * out = (hipComplex *) malloc(sizeof(hipComplex)*h*w);
	hipMalloc((void**)&d_input_image, sizeof(hipComplex)*w*h);
	hipMalloc((void**)&d_output_transform, sizeof(hipComplex)*w*h);
	hipMalloc((void**)&d_input_image_transpose, sizeof(hipComplex)*w*h);

	int grid_height = ceil((float)h/THREADS_PER_BLOCK_SIDE);
	int grid_width = ceil((float)h/THREADS_PER_BLOCK_SIDE);


	hipComplex h_input_image[w*h];
	for(int i = 0; i < w*h; i++) {
		h_input_image[i] = make_hipComplex(input_image[i].real, input_image[i].imag);
	}
	hipMemcpy(d_input_image, h_input_image, sizeof(hipComplex)*w*h, hipMemcpyHostToDevice);


	for(int p = 0; p < h; p++) {
		ufft_row(d_input_image + p*w, w);

	}

	hipMemcpy(out, d_input_image, sizeof(hipComplex)*w*h, hipMemcpyDeviceToHost);
	for(int i = 0; i < w; i++) {
		std::cout << out[i].x << ", ";
	}
	std::cout << std::endl;

	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image, d_input_image_transpose, w, h);

	for(int p = 0; p < h; p++) {
		ufft_row(d_input_image_transpose + p*w, w);
	}

	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image_transpose, d_output_transform, w, h);



	hipMemcpy(out, d_output_transform, sizeof(hipComplex)*w*h, hipMemcpyDeviceToHost);
//	for(int i = 0; i < w*h; i++) {
//		std::cout << out[i].x << ", ";
//	}
//	std::cout << std::endl;
//	hipMemcpy(h_input_image, d_output_transform, sizeof(hipComplex)*w*h,hipMemcpyDeviceToHost);
	for(int i =0; i < w*h; i++) {
		output_transform[i].real = out[i].x;
		output_transform[i].imag = out[i].y;
	}
	hipDeviceReset();
//	for(int i = 0; i < w*h; i++) {
//		std::cout << output_transform[i].real << ", ";
//	}
//	std::cout << std::endl;
}

