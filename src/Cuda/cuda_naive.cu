#include "hip/hip_runtime.h"
//
// Created by jbm on 12/5/18.
//


#include "cuda_naive.h"


__global__ void transpose(Complex * A, Complex * A_transpose, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < h && j < w)
		A_transpose[j * w + i] = A[i * w + j];

}

__global__ void idft_row(Complex * input, Complex * output, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;

	int this_element = i * w + j;


	if(i < h && j < w) {

		output[this_element].real = 0;
		output[this_element].imag = 0;

		for (int k = 0; k < w; k++) {
			double angle = 2 * M_PI * j * k / w;
			float W_real = cos(angle);

			float W_imag = sin(angle);
			int neighboring_element = i * w + k;
			output[this_element].real += (W_real * input[neighboring_element].real - W_imag * input[neighboring_element].imag);
			output[this_element].imag += (W_real * input[neighboring_element].imag + W_imag * input[neighboring_element].real);
		}
		output[this_element].real /= w;
		output[this_element].imag /= w;


	}
}

__global__ void dft_row(Complex * input, Complex * output, int w, int h) {
	int i = threadIdx.y + blockDim.y * blockIdx.y;
	int j = threadIdx.x + blockDim.x * blockIdx.x;

	int this_element = i * w + j;


	if(i < h && j < w) {

		output[this_element].real = 0;
		output[this_element].imag = 0;

		for (int k = 0; k < w; k++) {
			double angle = 2 * M_PI * j * k / w;
			float W_real = cos(angle);

			float W_imag = -sin(angle);
			int neighboring_element = i * w + k;
			output[this_element].real += (W_real * input[neighboring_element].real - W_imag * input[neighboring_element].imag);
			output[this_element].imag += (W_real * input[neighboring_element].imag + W_imag * input[neighboring_element].real);
		}


	}


}


void cuda_naive_forward(Complex * input_image, Complex * output_transform, InputImage input_image_meta) {
	Complex * d_input_image = NULL;
	Complex * d_output_transform = NULL;
	Complex * d_input_image_transpose = NULL;

	int w = input_image_meta.get_width();
	int h = input_image_meta.get_height();

	hipMalloc((void**)&d_input_image, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_output_transform, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_input_image_transpose, sizeof(Complex)*w*h);

	hipMemcpy(d_input_image, input_image, sizeof(Complex)*w*h, hipMemcpyHostToDevice);

	int grid_height = ceil((float)h/THREADS_PER_BLOCK_SIDE);
	int grid_width = ceil((float)h/THREADS_PER_BLOCK_SIDE);

	dft_row<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image, d_output_transform, w, h);
	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	dft_row<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image_transpose, d_output_transform, w, h);

	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	hipMemcpy(output_transform, d_input_image_transpose, sizeof(Complex)*w*h,hipMemcpyDeviceToHost);
}

void cuda_naive_inverse(Complex * input_image, Complex * output_transform, InputImage input_image_meta) {

	Complex * d_input_image = NULL;
	Complex * d_output_transform = NULL;
	Complex * d_input_image_transpose = NULL;

	int w = input_image_meta.get_width();
	int h = input_image_meta.get_height();

	hipMalloc((void**)&d_input_image, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_output_transform, sizeof(Complex)*w*h);
	hipMalloc((void**)&d_input_image_transpose, sizeof(Complex)*w*h);

	hipMemcpy(d_input_image, input_image, sizeof(Complex)*w*h, hipMemcpyHostToDevice);

	int grid_height = ceil((float)h/THREADS_PER_BLOCK_SIDE);
	int grid_width = ceil((float)h/THREADS_PER_BLOCK_SIDE);

	idft_row<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image, d_output_transform, w, h);
	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	idft_row<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_input_image_transpose, d_output_transform, w, h);

	transpose<<<dim3(grid_width,grid_height), dim3(THREADS_PER_BLOCK_SIDE, THREADS_PER_BLOCK_SIDE)>>>(d_output_transform, d_input_image_transpose, w, h);

	hipMemcpy(output_transform, d_input_image_transpose, sizeof(Complex)*w*h,hipMemcpyDeviceToHost);
}